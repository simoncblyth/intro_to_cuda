#include "hip/hip_runtime.h"
//  https://stackoverflow.com/questions/14927524/read-cudaarray-in-device-code
//  sample code from JackOLantern


#include <stdio.h>
#include <thrust/device_vector.h>

// --- 2D float texture
texture<float, hipTextureType2D, hipReadModeElementType> texRef;

// --- 2D surface memory
surface<void, 2> surf2D;


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/*************************************/
/* hipArray PRINTOUT TEXTURE KERNEL */
/*************************************/
__global__ void cudaArrayPrintoutTexture(int width, int height)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    printf("Thread index: (%i, %i); hipArray = %f\n", x, y, tex2D(texRef, x / (float)width + 0.5f, y / (float)height + 0.5f));
}

/*************************************/
/* hipArray PRINTOUT TEXTURE KERNEL */
/*************************************/
__global__ void cudaArrayPrintoutSurface(int width, int height)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    float temp;

    surf2Dread(&temp, surf2D, x * 4, y);

    printf("Thread index: (%i, %i); hipArray = %f\n", x, y, temp);
}

/********/
/* MAIN */
/********/
int main()
{
    int width = 3, height = 3;

    thrust::host_vector<float> h_data(width*height, 3.f);

    // --- Allocate CUDA array in device memory
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

    hipArray* cuArray;

    /*******************/
    /* TEXTURE BINDING */
    /*******************/
    gpuErrchk(hipMallocArray(&cuArray, &channelDesc, width, height));

    // --- Copy to host data to device memory
    gpuErrchk(hipMemcpyToArray(cuArray, 0, 0, thrust::raw_pointer_cast(h_data.data()), width*height*sizeof(float), hipMemcpyHostToDevice));

    // --- Set texture parameters
    texRef.addressMode[0] = hipAddressModeWrap;
    texRef.addressMode[1] = hipAddressModeWrap;
    texRef.filterMode = hipFilterModeLinear;
    texRef.normalized = true;

    // --- Bind the array to the texture reference
    gpuErrchk(hipBindTextureToArray(texRef, cuArray, channelDesc));

    // --- Invoking printout kernel
    dim3 dimBlock(3, 3);
    dim3 dimGrid(1, 1);
    cudaArrayPrintoutTexture<<<dimGrid, dimBlock>>>(width, height);

    gpuErrchk(hipUnbindTexture(texRef));

    gpuErrchk(hipFreeArray(cuArray));

    /******************/
    /* SURFACE MEMORY */
    /******************/
    gpuErrchk(hipMallocArray(&cuArray, &channelDesc, width, height, hipArraySurfaceLoadStore));

    // --- Copy to host data to device memory
    gpuErrchk(hipMemcpyToArray(cuArray, 0, 0, thrust::raw_pointer_cast(h_data.data()), width*height*sizeof(float), hipMemcpyHostToDevice));

    gpuErrchk(cudaBindSurfaceToArray(surf2D, cuArray));

    cudaArrayPrintoutSurface<<<dimGrid, dimBlock>>>(width, height);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipFreeArray(cuArray));

    return 0 ; 
}
