#include "hip/hip_runtime.h"
// ./SIMGStandaloneTest.sh

#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include <hip/hip_runtime.h>

#include <iostream>
#define SIMG_IMPLEMENTATION 1 
#include "SIMG.hh"

// https://stackoverflow.com/questions/14901491/cudamemcpytoarray/14929827#14929827

#include <stdio.h>
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)



__global__ void colorKernel(uchar4* output, hipTextureObject_t texObj, int width, int height, float theta) {
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    //if( x % 1000 == 0 ) printf("x %d y %d \n", x, y ); 
    //if( x == 1000 ) printf("x %d y %d \n", x, y ); 

	output[y * width + x] = make_uchar4( 255u, 0u, 0u, 255u ); 
}



__global__ void transformKernel(uchar4* output, hipTextureObject_t texObj, int width, int height, float theta) {
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	
    float u = x / (float) width;  // 0. -> 1. 
	float v = y / (float) height;

    // shift origin to center of image
	u -= 0.5f;                   //  -0.5 -> 0.5 
	v -= 0.5f;

    // rotate around the center
	float tu = u * cosf(theta) - v * sinf(theta) ;
	float tv = v * cosf(theta) + u * sinf(theta) ;

    // read from the texture  
    uchar4 c = tex2D<uchar4>(texObj, tu+0.5f, tv+0.5f); 

    //if( c.x != 0 ) printf(" c ( %d %d %d %d ) \n",c.x, c.y, c.z, c.w );  
    //c.x = 255u ; 
    c.w = 255u ; 

	output[y * width + x] = c ;
}

int main(int argc, char** argv)
{
    const char* ipath = argc > 1 ? argv[1] : "/tmp/i.png" ; 
    const char* opath = argc > 2 ? argv[2] : "/tmp/o.png" ; 

    int desired_channels = 4 ;  
    // hmm *desired_channels* does not change channels, the input image must be 4-channel 
    // (png are often 4-channel, jpg are 3 channel) 

    SIMG img(ipath, desired_channels); 
    std::cout << img.desc() << std::endl ; 
    assert( img.channels == 4 ); 

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar4>();

    hipArray *cuArray;
    hipMallocArray(&cuArray, &channelDesc, img.width, img.height );
    cudaCheckErrors("hipMallocArray");

    hipMemcpyToArray(cuArray, 0, 0, img.data, img.width*img.height*4*sizeof(unsigned char), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpyToArray");

	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;

    // https://docs.nvidia.com/cuda/cuda-runtime-api/structcudaTextureDesc.html
	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;

	//texDesc.filterMode = hipFilterModeLinear;
	texDesc.filterMode = hipFilterModePoint;    // switch off interpolation, as that gives error with non-float texture  

	texDesc.readMode = hipReadModeElementType;  // return data of the type of the underlying buffer
	texDesc.normalizedCoords = 1 ;            // addressing into the texture with floats in range 0:1

	// Create texture object
	hipTextureObject_t texObj = 0;
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

	// Allocate result of transformation in device memory
	uchar4* d_output;
	hipMalloc(&d_output, img.width * img.height * 4*sizeof(unsigned char));

	dim3 dimBlock(16, 16);
	dim3 dimGrid((img.width + dimBlock.x - 1) / dimBlock.x, (img.height + dimBlock.y - 1) / dimBlock.y);

    float theta = 1.f ; 

	//colorKernel<<<dimGrid, dimBlock>>>(d_output, texObj, img.width, img.height, theta );
	transformKernel<<<dimGrid, dimBlock>>>(d_output, texObj, img.width, img.height, theta );
    hipDeviceSynchronize();      
    cudaCheckErrors("hipDeviceSynchronize"); 
    // Fatal error: hipDeviceSynchronize (linear filtering not supported for non-float type at SIMGStandaloneTest.cu:123)


    uchar4* output = new uchar4[img.width*img.height] ; 
    hipMemcpy(output, d_output, img.width*img.height*sizeof(uchar4), hipMemcpyDeviceToHost);     

    std::cout << "writing to " << opath << std::endl ; 

    SIMG img2(img.width, img.height, img.channels, (unsigned char*)output ); 
    img2.writePNG(opath); 

    hipDeviceSynchronize();  

	hipDestroyTextureObject(texObj);
	hipFreeArray(cuArray);

    delete[] output ; 
	hipFree(d_output);

    return 0;
}

