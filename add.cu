// nvcc add.cu -run && rm a.out


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>


__global__ void add(int *a, int *b, int *c) 
{ 
    *c = *a + *b;
}



int main(void) 
{
  int a, b, c; // host 
  int *d_a, *d_b, *d_c; // pointers to device memory 

  int size = sizeof(int);
 
   // Allocate space for device copies of a, b, c
   hipMalloc((void **)&d_a, size); 
   hipMalloc((void **)&d_b, size); 
   hipMalloc((void **)&d_c, size);
      

   // Setup input values
   a = 2; 
   b = 7;

   int c_expect = a + b ; 
 

   // Copy inputs to device
   hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice); 
   hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

   // Launch add() kernel on GPU
   add<<<1,1>>>(d_a, d_b, d_c);

   // Copy result back to host
   hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);


   printf(" a %d b %d c %d  \n", a, b, c );

   assert( c == c_expect );  
   

   // Cleanup
   hipFree(d_a); 
   hipFree(d_b); 
   hipFree(d_c); 
   return 0;

}


